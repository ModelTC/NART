#include "hip/hip_runtime.h"
/*
 * Copyright 2022 SenseTime Group Limited
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "art/log.h"
#include "art/module.h"
#include "art/op.h"
#include "art/op_settings.h"
#include "art/op_tp.h"

#include "../cuda_workspace.h"

#ifdef __cplusplus
extern "C" {
#endif

typedef struct {
    op_t o;
    bool share;
} op_prelu_t;

op_prelu_t *op_cuda_prelu_tp_alloc(workspace_t *ws)
{
    (void)ws;
    op_prelu_t *res = (op_prelu_t *)malloc(sizeof(op_prelu_t));
    memset(res, 0, sizeof(op_prelu_t));
    return res;
}

void op_cuda_prelu_tp_config(op_t *op)
{
    CHECK(op_setting_single_get(op, SETTING_PRELU_SHARE, dtBOOL, &((op_prelu_t *)op)->share));
}

void op_cuda_prelu_tp_destroy(op_t *op) { (void)op; }

void op_cuda_prelu_tp_dealloc(op_t *op)
{
    if (NULL != op)
        free(op);
}

__global__ void op_cuda_prelu_share_kernel(
    float *dst, const float *src, const float slope, const int channel, const int size)
{
    CUDA_KERNEL_LOOP(index, size)
    {
        for (int c = 0; c < channel; ++c) {
            dst[index + c * size]
                = fmaxf(0.f, src[index + c * size]) + slope * fminf(0.f, src[index + c * size]);
        }
    }
}

__global__ void op_cuda_prelu_kernel(
    float *dst, const float *src, const float *slopes, const int channel, const int size)
{
    CUDA_KERNEL_LOOP(index, size)
    {
        for (int c = 0; c < channel; ++c) {
            dst[index + c * size]
                = fmaxf(0.f, src[index + c * size]) + slopes[c] * fminf(0.f, src[index + c * size]);
        }
    }
}

static void op_cuda_prelu_run(op_t *op)
{
    size_t count = shape_count(&op->output_tensors[0].shape);
    int n = op->output_tensors[0].shape.dim[0];
    int channel = op->output_tensors[0].shape.dim[1];
    int size = count / n / channel;
    const float *input_0 = (float *)mem_data(op->input_tensors[0]->mem);
    float *output_0 = (float *)mem_data(op->output_tensors[0].mem);
    bool share = ((op_prelu_t *)op)->share;
    if (share) {
        float slope = ((float *)mem_cpu_data(op->input_tensors[1]->mem))[0];
        op_cuda_prelu_share_kernel<<<
            (count + 1024 - 1) / 1024, 1024, 0, CUDA_WORKSPACE_STREAM(op->workspace)>>>(
            output_0, input_0, slope, channel, size);
    } else {
        const float *input_1 = (float *)mem_data(op->input_tensors[1]->mem);
        op_cuda_prelu_kernel<<<
            (count + 1024 - 1) / 1024, 1024, 0, CUDA_WORKSPACE_STREAM(op->workspace)>>>(
            output_0, input_0, input_1, channel, size);
    }
}

void op_cuda_prelu_tp_prepare(op_t *op)
{
    int i;
    for (i = 0; i < op->input_size; ++i) {
        tensor_alloc(op->input_tensors[i]);
    }
    for (i = 0; i < op->output_size; ++i) {
        tensor_alloc(&op->output_tensors[i]);
    }
    switch (op->input_tensors[0]->dtype) {
    case dtFLOAT32:
        op->run_func = op_cuda_prelu_run;
        break;
    default:
        CHECK(false);
        break;
    }
}

#ifdef __cplusplus
}
#endif
