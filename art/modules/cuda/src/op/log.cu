#include "hip/hip_runtime.h"
/*
 * Copyright 2022 SenseTime Group Limited
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdlib.h>
#include <string.h>

#include "art/log.h"
#include "art/module.h"
#include "art/op.h"
#include "art/op_tp.h"

#include "../cuda_workspace.h"

#ifdef __cplusplus
extern "C" {
#endif

typedef struct {
    op_t o;
} op_log_t;

op_log_t *op_cuda_log_tp_alloc(workspace_t *ws)
{
    (void)ws;
    op_log_t *res = (op_log_t *)malloc(sizeof(op_log_t));
    memset(res, 0, sizeof(op_log_t));
    return res;
}

void op_cuda_log_tp_config(op_t *op) { (void)op; }

void op_cuda_log_tp_destroy(op_t *op) { (void)op; }

void op_cuda_log_tp_dealloc(op_t *op)
{
    if (NULL != op)
        free(op);
}

__global__ void op_cuda_log_kernel(float *c, const float *a, size_t size)
{
    CUDA_KERNEL_LOOP(i, size) { c[i] = logf(a[i]); }
}

static void op_cuda_log_run(op_t *op)
{
    size_t count = shape_count(&op->output_tensors[0].shape);
    const float *input_0 = (const float *)mem_data(op->input_tensors[0]->mem);
    float *output_0 = (float *)mem_data(op->output_tensors[0].mem);
    op_cuda_log_kernel<<<
        (count + 1024 - 1) / 1024, 1024, 0, CUDA_WORKSPACE_STREAM(op->workspace)>>>(
        output_0, input_0, count);
}

void op_cuda_log_tp_prepare(op_t *op)
{
    int i;
    for (i = 0; i < op->input_size; ++i) {
        tensor_alloc(op->input_tensors[i]);
    }
    for (i = 0; i < op->output_size; ++i) {
        tensor_alloc(&op->output_tensors[i]);
    }
    switch (op->input_tensors[0]->dtype) {
    case dtFLOAT32:
        op->run_func = op_cuda_log_run;
        break;
    default:
        CHECK(false);
        break;
    }
}

#ifdef __cplusplus
}
#endif
